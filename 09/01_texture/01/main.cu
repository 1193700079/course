#include "hip/hip_runtime.h"
#include <cstdio>
#include <vector>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include "hip/hip_vector_types.h"
#include "CudaAllocator.cuh"
#include "CudaArray.cuh"
#include "ticktock.h"
#include "writevdb.h"

__global__ void kernel(hipTextureObject_t texVel, hipSurfaceObject_t sufLoc, unsigned int n) {
    unsigned int x = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int y = threadIdx.y + blockDim.y * blockIdx.y;
    unsigned int z = threadIdx.z + blockDim.z * blockIdx.z;
    if (x >= n || y >= n || z >= n) return;
    float4 vel = tex3D<float4>(texVel, x, y, z);
    float4 loc = make_float4(x + 0.5f, y + 0.5f, z + 0.5f, 1.f) - vel;
    surf3Dwrite<float4>(loc, sufLoc, x, y, z, hipBoundaryModeTrap);
}

int main() {
    unsigned int n = 2;

    auto arrLoc = CudaArray<float4>::make({{n, n, n}});
    auto sufLoc = CudaSurface<float4>::make(arrLoc);
    auto arrVel = CudaArray<float4>::make({{n, n, n}});
    auto texVel = CudaTexture<float4>::make(arrVel);

    std::vector<float4> cpuVel(n * n * n);
    for (int z = 0; z < n; z++) {
        for (int y = 0; y < n; y++) {
            for (int x = 0; x < n; x++) {
                cpuVel[x + n * (y + n * z)] = make_float4(1.f, 0.f, 0.f, 0.f);
            }
        }
    }
    arrVel.copyIn(cpuVel.data());

    kernel<<<dim3((n + 7) / 8, (n + 7) / 8, (n + 7) / 8), dim3(8, 8, 8)>>>(texVel.get(), sufLoc.get(), n);

    std::vector<float4> cpuLoc(n * n * n);
    arrLoc.copyOut(cpuLoc.data());
    for (int z = 0; z < n; z++) {
        for (int y = 0; y < n; y++) {
            for (int x = 0; x < n; x++) {
                float4 val = cpuLoc[x + n * (y + n * z)];
                printf("%d,%d,%d: %f,%f,%f,%f\n", x, y, z, val.x, val.y, val.z, val.w);
            }
        }
    }
    //writevdb<float, 3>("/tmp/a.vdb", n, n, n, arr.data());

    return 0;
}
