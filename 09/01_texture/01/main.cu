#include "hip/hip_runtime.h"
#include <cstdio>
#include <vector>
#include <memory>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include "CudaAllocator.h"
#include "ticktock.h"

struct DisableCopy {
    DisableCopy() = default;
    DisableCopy(DisableCopy const &) = delete;
    DisableCopy &operator=(DisableCopy const &) = delete;
    DisableCopy(DisableCopy &&) = delete;
    DisableCopy &operator=(DisableCopy &&) = delete;
};

template <class T>
class CudaArray {
    struct BuildArgs {
        std::array<unsigned int, 3> const dim{};
        int flags = 0; // or hipArraySurfaceLoadStore
    };

    struct Impl : DisableCopy {
        hipArray *m_cuArray{};
        std::array<unsigned int, 3> m_dim{};

        explicit Impl(BuildArgs _args) : m_dim(_args.dim) {
            hipChannelFormatDesc channelDesc = hipCreateChannelDesc<T>();  // or hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned)
            checkCudaErrors(hipMalloc3DArray(&m_cuArray, &channelDesc, make_hipExtent(m_dim[0], m_dim[1], m_dim[2]), _args.flags));
        }

        void assign(T *_data) {
            hipMemcpy3DParms copy3DParams{};
            copy3DParams.srcPtr = make_hipPitchedPtr((void *)_data, m_dim[0] * sizeof(T), m_dim[1], m_dim[2]);
            copy3DParams.dstArray = m_cuArray;
            copy3DParams.extent = make_hipExtent(m_dim[0], m_dim[1], m_dim[2]);
            copy3DParams.kind = hipMemcpyHostToDevice;
            checkCudaErrors(hipMemcpy3D(&copy3DParams));
        }

        ~Impl() {
            checkCudaErrors(hipFreeArray(m_cuArray));
        }
    };

    std::shared_ptr<Impl> impl;

public:
    explicit CudaArray(BuildArgs _args) : impl(std::make_shared<Impl>(_args)) {
    }

    CudaArray &assign(T *_data) const {
        impl->assign(_data);
        return *this;
    }

    operator hipArray *() const {
        return impl->m_cuArray;
    }
};

template <class T>
class CudaSurface {
    struct Impl : DisableCopy {
        hipSurfaceObject_t m_cuSuf{};
        CudaArray<T> m_cuarr;

        explicit Impl(CudaArray<T> _cuarr) : m_cuarr(_cuarr) {
            hipResourceDesc resDesc{};
            resDesc.resType = hipResourceTypeArray;

            resDesc.res.array.array = m_cuarr;
            hipCreateSurfaceObject(&m_cuSuf, &resDesc);
        }

        ~Impl() {
            checkCudaErrors(hipDestroySurfaceObject(m_cuSuf));
        }
    };

    std::shared_ptr<Impl> impl;

public:
    explicit CudaSurface(CudaArray<T> _cuarr) : impl(std::make_shared<Impl>(_cuarr)) {
    }

    CudaArray<T> &getArray() const {
        return impl->m_cuarr;
    }

    operator hipSurfaceObject_t() const {
        return impl->m_cuSuf;
    }
};

template <class T>
class CudaTexture {
    struct Impl : DisableCopy {
        hipTextureObject_t m_cuTex{};
        CudaArray<T> m_cuarr;

        explicit Impl(CudaArray<T> _cuarr) : m_cuarr(_cuarr) {
            hipResourceDesc resDesc{};
            resDesc.resType = hipResourceTypeArray;
            resDesc.res.array.array = m_cuarr;

            hipTextureDesc texDesc{};
            texDesc.addressMode[0] = hipAddressModeClamp; // or hipAddressModeWrap
            texDesc.addressMode[1] = hipAddressModeClamp; // or hipAddressModeWrap
            texDesc.addressMode[2] = hipAddressModeClamp; // or hipAddressModeWrap
            texDesc.filterMode = hipFilterModePoint;      // or hipFilterModeLinear
            texDesc.readMode = hipReadModeElementType;    // or hipReadModeNormalizedFloat
            texDesc.normalizedCoords = false;              // or true

            checkCudaErrors(hipCreateTextureObject(&m_cuTex, &resDesc, &texDesc, NULL));
        }

        ~Impl() {
            checkCudaErrors(hipDestroyTextureObject(m_cuTex));
        }
    };

    std::shared_ptr<Impl> impl;

public:
    explicit CudaTexture(CudaArray<T> _cuarr) : impl(std::make_shared<Impl>(_cuarr)) {
    }

    CudaArray<T> &getArray() const {
        return impl->m_cuarr;
    }

    operator hipTextureObject_t() const {
        return impl->m_cuTex;
    }
};

__global__ void kernel(hipSurfaceObject_t out, hipTextureObject_t in) {
    int x = 0, y = 0;
    float fx = 0, fy = 0, fz = 0;
    float value = tex3D<float>(in, fx, fy, fz);
    value += 1;
    surf2Dwrite(value, out, x, y);
    // or hipBoundaryModeTrap, hipBoundaryModeClamp
}

int main() {
    CudaSurface<float> out(CudaArray<float>({{1, 1, 1}, hipArraySurfaceLoadStore}));
    CudaTexture<float> in(CudaArray<float>({{1, 1, 1}, 0}));
    return 0;
}
